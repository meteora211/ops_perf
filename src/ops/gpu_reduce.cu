#include "hip/hip_runtime.h"
#include "gpu_reduce.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

double reduce_cuda_executor(void(*cuda_func)(float *, float *, const int),
                   std::shared_ptr<float[]> lhs,
                   std::shared_ptr<float[]> res,
                   int N,
                   dim3 grid,
                   dim3 block) {
  size_t lhs_size = N * sizeof(float);
  // reduce 返回numBlocks个数，后续可以在cpu或者重新使用1个block reduce返回值
  size_t res_size = block * sizeof(float);

  float* lhs_device, *res_device;
  hipMalloc(&lhs_device, lhs_size);
  hipMalloc(&res_device, res_size);

  hipMemcpy(lhs_device, lhs.get(), lhs_size, hipMemcpyHostToDevice);
  hipMemcpy(res_device, res.get(), res_size, hipMemcpyHostToDevice);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  cuda_func<<<grid, block>>>(lhs_device, res_device, N);

  hipEventRecord(end);
  hipEventSynchronize(end);

  float msec;
  hipEventElapsedTime(&msec, start, end);

  hipMemcpy(res.get(), res_device, res_size, hipMemcpyDeviceToHost);


  hipFree(lhs_device);
  hipFree(res_device);
  return msec;
}

__global__ void reduce_naive(float* lhs, float* res, int N) {
    __shared__ float sdata[THREAD_PER_BLOCK];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    sdata[tid] = lhs[i];
    __syncthreads();
    
    // reduce sdata
    for (int i = 1; i < blockDim.x; i*=2) {
        // int stride = i;
        if (tid % (i * 2) == 0) {
            sdata[tid] += sdata[tid + i];
        }
        __syncthreads();
    }
    
    if (tid==0) res[blockIdx.x] = sdata[0];
}

double reduce_cuda_naive(std::shared_ptr<float[]> lhs, std::shared_ptr<float[]> res, int N) {
  const int BN = 32;

  dim3 grid((N + BN - 1) / BN);
  dim3 block(BN);
  return reduce_cuda_executor(reduce_naive, lhs, res, N, grid, block);
}