
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <memory>
#include <chrono>
#include <math.h>
#include <string>

#define BLOCK_SIZE 16

struct CUProfiler {
  CUProfiler(std::string name) : name(name) {
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
  }

  ~CUProfiler() {
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipEventElapsedTime(&msec, start, end);
    std::cout << "[" << name << "]: " << "Average elasped time: " << msec << " ms." << std::endl;
  }
  std::string name;
  float msec;
  hipEvent_t start;
  hipEvent_t end;
};

void fullfill_rand(float* input, int nelm) {
  std::srand(std::time(nullptr)); // use current time as seed for random generator
  for (int i = 0; i < nelm; ++i) {
    input[i] = std::rand() / static_cast<float>(RAND_MAX);
  }
}

bool matrixChecker(float* res, float* expect, int M, int N) {
  bool correct = true;
  // test relative error by the formula
  //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
  double eps = 1.e-6; // machine zero

  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      int index = i * N + j;
      double abs_err = fabs(res[index] - expect[index]);
      double dot_length = M * N;
      double abs_val = fabs(expect[index]);
      double rel_err = abs_err / abs_val / dot_length;

      if (rel_err > eps) {
          correct = false;
      }
    }
  }

  printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");
  return correct;
}

void matmulBase(float* lhs, float* rhs, float* res, int M, int N, int K) {
  // lhs(M*K) * rhs(K*N) = res(M*N)
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      float sum = 0;
      for (int k = 0; k < K; ++k) {
        // sum += lhs[i, k] * rhs[k, j];
        sum += lhs[i * K + k] * rhs[k * N + j];
      }
      res[i * N + j] = sum;
    }
  }
}


__global__ void matmulNaive(float* A, float* B, float* C, int M, int N, int K) {
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (i < M && j < N) {
    float sum = 0;
    for (int k = 0; k < K; ++k) {
      sum += A[i * M + k] * B[k * N + j];
    }
    C[i * M + j] = sum;
  }
}

__global__ void matmulShared(float* A, float* B, float* C, int M, int N, int K) {
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int bi = threadIdx.x;
  int bj = threadIdx.y;
  
  if (i < M && j < N) {
    float sum = 0;
    __shared__ float shareA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shareB[BLOCK_SIZE][BLOCK_SIZE];
    for (int k = 0; k < K; k+=BLOCK_SIZE) {
      shareA[bi][bj] = A[i * M + k];
      shareB[bj][bi] = B[k * N + j];
      __syncthreads();
      for (int inner = 0; inner < BLOCK_SIZE; ++inner) {
        sum += shareA[bi][inner] * shareB[bj][inner];
      }
      __syncthreads();
    }
    C[i * M + j] = sum;
  }
}

__global__ void matmulBankOpt(float* A, float* B, float* C, int M, int N, int K) {
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int bi = threadIdx.x;
  int bj = threadIdx.y;
  
  if (i < M && j < N) {
    float sum = 0;
    __shared__ float shareA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shareB[BLOCK_SIZE][BLOCK_SIZE];
    for (int k = 0; k < K; k+=BLOCK_SIZE) {
      shareA[bj][bi] = A[i * M + k];
      shareB[bi][bj] = B[k * N + j];
      __syncthreads();
      for (int inner = 0; inner < BLOCK_SIZE; ++inner) {
        sum += shareA[inner][bi] * shareB[inner][bj];
      }
      __syncthreads();
    }
    C[i * M + j] = sum;
  }
}

__global__ void matmulBankOpt2(float* A, float* B, float* C, int M, int N, int K) {
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int bi = threadIdx.x;
  int bj = threadIdx.y;
  
  if (i < M && j < N) {
    float sum = 0;
    __shared__ float shareA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shareB[BLOCK_SIZE][BLOCK_SIZE + 1];
    for (int k = 0; k < K; k+=BLOCK_SIZE) {
      shareA[bj][bi] = A[i * M + k];
      shareB[bi][bj] = B[k * N + j];
      __syncthreads();
      for (int inner = 0; inner < BLOCK_SIZE; ++inner) {
        sum += shareA[inner][bi] * shareB[inner][bj];
      }
      __syncthreads();
    }
    C[i * M + j] = sum;
  }
}

int main() {
  std::cout << "test matmul naive" << std::endl;
  int M = 1024;
  int N = 3000;
  int K = 2000;
  const bool checkResult = false;
  const int iteration = 50;
  
  float* hA = static_cast<float*>(malloc(M * K * sizeof(float)));
  float* hB = static_cast<float*>(malloc(K * N * sizeof(float)));
  float* hC = static_cast<float*>(malloc(M * N * sizeof(float)));
  
  float* dA;
  hipMalloc(&dA, M * K * sizeof(float));
  float* dB;
  hipMalloc(&dB, K * N * sizeof(float));
  float* dC;
  hipMalloc(&dC, M * N * sizeof(float));
  
  hipMemcpy(hA, dA, M * K * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(hB, dB, K * N * sizeof(float), hipMemcpyHostToDevice);
  
  dim3 threadPerBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 blockPerGrid((M + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
  // matmulBankOpt2<<<blockPerGrid, threadPerBlock>>>(dA, dB, dC, M, N, K);
  // warm up
  for (int i = 0; i < 20; ++i) {
    matmulNaive<<<blockPerGrid, threadPerBlock>>>(dA, dB, dC, M, N, K);
  }
  {
    CUProfiler profiler("naive");
    for (int i = 0; i < iteration; ++i) matmulNaive<<<blockPerGrid, threadPerBlock>>>(dA, dB, dC, M, N, K);
  }
  {
    CUProfiler profiler("shared mem");
    for (int i = 0; i < iteration; ++i) matmulShared<<<blockPerGrid, threadPerBlock>>>(dA, dB, dC, M, N, K);
  }
  {
    CUProfiler profiler("bank opt");
    for (int i = 0; i < iteration; ++i) matmulBankOpt<<<blockPerGrid, threadPerBlock>>>(dA, dB, dC, M, N, K);
  }
  {
    CUProfiler profiler("bank opt2");
    for (int i = 0; i < iteration; ++i) matmulBankOpt2<<<blockPerGrid, threadPerBlock>>>(dA, dB, dC, M, N, K);
  }

  hipMemcpy(hC, dC, M * N * sizeof(float), hipMemcpyDeviceToHost);
  if (checkResult) {
    float* hRef = static_cast<float*>(malloc(M * N * sizeof(float)));
    matmulBase(hA, hB, hRef, M, N, K);
    matrixChecker(hRef, hC, M, N);
  }
}